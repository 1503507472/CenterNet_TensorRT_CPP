#include "hip/hip_runtime.h"
//
// Created by StephenFang on 2019/12/25.
//

#include <ct_trt_config.h>

__device__ float Logist(float data){ return 1./(1. + exp(-data)); }

///检测前向
//CTdetforward_gpu(static_cast<const float *>(mCudaBuffers[1]),static_cast<const float *>(mCudaBuffers[2]),
//static_cast<const float *>(mCudaBuffers[3]),static_cast<float *>(cudaOutputBuffer),
//        ouputSize,ouputSize,classNum,kernelSize,visThresh)
__global__ void CTdetforward_kernel(const float *hm, const float *reg,const float *wh ,
                                    float *output,const int w,const int h,const int classes,const int kernerl_size,const float visthresh  ) {

    int idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x; //当前thread位置
    if (idx >= w*h) return; //?
    int padding = kernerl_size/2; //kernel_size default = 4
    int offset = - padding /2;
    int stride = w*h; //计算跨步
    int grid_x = idx % w ; //取余数
    int grid_y = idx / w ; //取整数
    int cls,l,m;
    float c_x,c_y;
    for (cls = 0; cls < classes; ++cls ) //遍历每一个类别？
    {
        int objIndex = stride * cls + idx; //物件的位置？？
        float objProb = hm[objIndex]; //求出当前类的概率？
        float max=-1;
        int max_index =0;

        for(l=0 ;l < kernerl_size ; ++l) //kernel_size default = 4
            for(m=0 ; m < kernerl_size ; ++m){
                int cur_x = offset + l + grid_x;
                int cur_y = offset + m + grid_y;
                int cur_index = cur_y * w + cur_x + stride*cls;
                int valid = (cur_x>=0 && cur_x < w && cur_y >=0 && cur_y <h );
                float val = (valid !=0 ) ? Logist(hm[cur_index]): -1;
                max_index = (val > max) ? cur_index : max_index;
                max = (val > max ) ?  val: max ;
            }
        objProb = Logist(objProb);
        if((max_index == objIndex) && (objProb > visthresh)){

            int resCount = (int)atomicAdd(output,1);
            //printf("%d",resCount);
            char* data = (char * )output + sizeof(float) + resCount*sizeof(Detection);
            Detection* det =  (Detection*)(data);
            c_x = grid_x + reg[idx] ; c_y  = grid_y + reg[idx+stride];

            ///计算出bbox的坐标
            det->bbox.x1 = (c_x - wh[idx]/2)*4;
            det->bbox.y1 = (c_y - wh[idx+stride]/2)*4 ;
            det->bbox.x2 = (c_x + wh[idx]/2)*4;
            det->bbox.y2 = (c_y + wh[idx+stride]/2)*4;
            det->classId = cls;
            det->prob = objProb;
        }
    }
}

__global__ void CTfaceforward_kernel(const float *hm, const float *wh,const float *reg,const float* landmarks,
                                     float *output,const int w,const int h,const int classes,const int kernerl_size,const float visthresh  ) {
    int idx = (blockIdx.x + blockIdx.y * gridDim.x) * blockDim.x + threadIdx.x;
    if (idx >= w*h) return;
    int padding = kernerl_size/2;
    int offset = - padding /2;
    int stride = w*h;
    int grid_x = idx % w ;
    int grid_y = idx / w ;
    int cls,l,m,mark_id;
    float c_x,c_y,scale_w,scale_h;
    for (cls = 0; cls < classes; ++cls )
    {
        int objIndex = stride * cls + idx;
        float objProb = hm[objIndex];
        float max=-1;
        int max_index =0;
        for(l=0 ;l < kernerl_size ; ++l)
            for(m=0 ; m < kernerl_size ; ++m){
                int cur_x = offset + l + grid_x;
                int cur_y = offset + m + grid_y;
                int cur_index = cur_y * w + cur_x + stride*cls;
                int valid = (cur_x>=0 && cur_x < w && cur_y >=0 && cur_y <h );
                float val = (valid !=0 ) ? hm[cur_index]: -1;
                max_index = (val > max) ? cur_index : max_index;
                max = (val > max ) ?  val: max ;
            }
        //printf("%f\n",objProb);
        if((max_index == objIndex) && (objProb > visthresh)){

            int resCount = (int)atomicAdd(output,1);
            //printf("%d",resCount);
            char* data = (char * )output + sizeof(float) + resCount*sizeof(Detection);
            Detection* det =  (Detection*)(data);
            c_x = (grid_x + reg[idx+stride] + 0.5)*4 ; c_y  = (grid_y + reg[idx] + 0.5) * 4;
            scale_w =  expf(wh[idx+stride]) * 4 ; scale_h  = expf(wh[idx]) * 4;
            det->bbox.x1 = c_x - scale_w/2;
            det->bbox.y1 = c_y - scale_h/2 ;
            det->bbox.x2 = c_x + scale_w/2;
            det->bbox.y2 = c_y + scale_h/2;
            det->prob = objProb;
            det->classId = cls;
            for(mark_id=0 ; mark_id < 5 ; ++mark_id ){
                det->marks[mark_id].x = det->bbox.x1 + landmarks[idx + (2*mark_id+1)*stride]*scale_w;
                det->marks[mark_id].y = det->bbox.y1 + landmarks[idx + (2*mark_id)*stride]*scale_h;
            }
        }
    }
}

///把前面定义的kernel 交给GPU运行
void CTdetforward_gpu(const float *hm, const float *reg,const float *wh ,float *output,
                      const int w,const int h,const int classes,const int kernerl_size, const float visthresh ){
    uint num = w * h;
    CTdetforward_kernel<<<cudaGridSize(num),BLOCK>>>(hm,reg,wh,output,w,h,classes,kernerl_size,visthresh);
    //执行GPU kernel
}



void CTfaceforward_gpu(const float *hm, const float *wh,const float *reg,const float* landmarks,float *output,
                       const int w,const int h,const int classes,const int kernerl_size, const float visthresh ){
    uint num = w * h;
    CTfaceforward_kernel<<<cudaGridSize(num),BLOCK>>>(hm,wh,reg,landmarks,output,w,h,classes,kernerl_size,visthresh);
}
